
#include <hip/hip_runtime.h>
__global__ void
exemple(void){
  int identifiant_local = threadIdx.x;
  int identifiant_global = blockIdx.x * blockDim.x + threadIdx.x;
}

int
main(){
  exemple<<<512,512>>>();
  return 0;
}
